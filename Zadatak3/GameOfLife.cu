#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "GameOfLife.cuh"

namespace GameOfLifeCUDALibrary {

	__global__ void GameOfLife(const unsigned char* lifeData, const int worldWidth, const int worldHeight, unsigned char* resultLifeData)
	{
		int worldSize = worldWidth * worldHeight;
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (int i = index; i < worldSize; i += stride)
		{
			int x = i % worldWidth;
			int yAbs = i - x;

			int xLeft = (x + worldWidth - 1) % worldWidth;
			int xRight = (x + 1) % worldWidth;

			int yAbsUp = (yAbs + worldSize - worldHeight) % worldSize;
			int yAbsDown = (yAbs + worldHeight) % worldSize;

			//Count alive cells
			int aliveCells = lifeData[xLeft + yAbsUp] + lifeData[x + yAbsUp] + lifeData[xRight + yAbsUp]
				+ lifeData[xLeft + yAbs] + lifeData[xRight + yAbs]
				+ lifeData[xLeft + yAbsDown] + lifeData[x + yAbsDown] + lifeData[xRight + yAbsDown];

			resultLifeData[x + yAbs] = aliveCells == 3 || (aliveCells == 2 && lifeData[x + yAbs]) ? 1 : 0;
		}
	}

	
	__global__ void InitWorld(unsigned char* lifeData, int worldWidth, int worldHeight, unsigned char* resultLifeData)
	{
		int worldSize = worldWidth * worldHeight;
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (int i = index; i < worldSize; i += stride)
		{
			lifeData[i] = 0;
			resultLifeData[i] = 0;
		}
	}

	__global__ void InitWithImage(int imageWidth, int imageHeight, const unsigned char* imageBuffer, int worldWidth, int worldHeight,
		unsigned char* lifeData)
	{
		int imageSize = imageWidth * imageHeight;
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (int i = index; i < imageSize; i += stride)
		{
			int xImage = i % imageWidth;
			int yImage = (i - xImage) / imageWidth;

			lifeData[yImage * worldWidth + xImage] = imageBuffer[xImage + yImage*imageWidth] == 255 ? 1 : 0;
		}
	}

	__global__ void WriteImage(int imageWidth, int imageHeight, unsigned char* imageBuffer, int worldWidth, int worldHeight,
		unsigned char* lifeData)
	{
		int imageSize = imageWidth * imageHeight;
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (int i = index; i < imageSize; i += stride)
		{
			int xImage = i % imageWidth;
			int yImage = (i - xImage) / imageWidth;
	
			imageBuffer[xImage + yImage * imageWidth] = lifeData[yImage * worldWidth + xImage] == 1 ? 255 : 0;
		}
	}

	__global__ void WriteRGBImage(unsigned char* lifeData, int worldWidth, int worldHeight, Pixel* image)
	{
		int imageSize = worldWidth * worldHeight;
		int index = blockIdx.x * blockDim.x + threadIdx.x;
		int stride = blockDim.x * gridDim.x;

		for (int i = index; i < imageSize; i += stride)
		{
			int x = i % worldWidth;
			int yAbs = i - x;

			if (lifeData[x + yAbs] == 1)
			{
				image[x + yAbs].r = 255;
				image[x + yAbs].b = 255;
				image[x + yAbs].g = 255;

				int xLeft = (x + worldWidth - 1) % worldWidth;
				int xRight = (x + 1) % worldWidth;
				int xLeft2 = (x + worldWidth - 2) % worldWidth;
				int xRight2 = (x + 2) % worldWidth;

				int yAbsUp = (yAbs + imageSize - worldHeight) % imageSize;
				int yAbsDown = (yAbs + worldHeight) % imageSize;
				int yAbsUp2 = (yAbs + imageSize - 2 * worldHeight) % imageSize;
				int yAbsDown2 = (yAbs + 2 * worldHeight) % imageSize;

				if (lifeData[x + yAbs] == 1 &&
					lifeData[xLeft + yAbs] == 1 &&
					lifeData[xRight + yAbs] == 1 &&
					lifeData[xLeft + yAbsUp] == 0 &&
					lifeData[x + yAbsUp] == 0 &&
					lifeData[xRight + yAbsUp] == 0 &&
					lifeData[xLeft + yAbsDown] == 0 &&
					lifeData[x + yAbsDown] == 0 &&
					lifeData[xRight + yAbsDown] == 0 &&
					lifeData[xLeft2 + yAbs] == 0 &&
					lifeData[xLeft2 + yAbsUp] == 0 &&
					lifeData[xLeft2 + yAbsDown] == 0 &&
					lifeData[xRight2 + yAbs] == 0 &&
					lifeData[xRight2 + yAbsUp] == 0 &&
					lifeData[xRight2 + yAbsDown] == 0)
				{
					image[x + yAbs].b = 0;
					image[xLeft + yAbs].b = 0;
					image[xRight + yAbs].b = 0;
					image[x + yAbs].g = 0;
					image[xLeft + yAbs].g = 0;
					image[xRight + yAbs].g = 0;

				}
				else if (lifeData[x + yAbs] == 1 &&
					lifeData[x + yAbsUp] == 1 &&
					lifeData[x + yAbsDown] == 1 &&
					lifeData[xLeft + yAbs] == 0 &&
					lifeData[xLeft + yAbsUp] == 0 &&
					lifeData[xLeft + yAbsDown] == 0 &&
					lifeData[xRight + yAbs] == 0 &&
					lifeData[xRight + yAbsUp] == 0 &&
					lifeData[xRight + yAbsDown] == 0 &&
					lifeData[xLeft + yAbsUp2] == 0 &&
					lifeData[xLeft + yAbsDown2] == 0 &&
					lifeData[xRight + yAbsUp2] == 0 &&
					lifeData[xRight + yAbsDown2] == 0)
				{
					image[x + yAbs].g = 0;
					image[x + yAbsUp].g = 0;
					image[x + yAbsDown].g = 0;
					image[x + yAbs].b = 0;
					image[x + yAbsUp].b = 0;
					image[x + yAbsDown].b = 0;
				}
			}
			else {
				image[x + yAbs].r = 0;
				image[x + yAbs].b = 0;
				image[x + yAbs].g = 0;
			}
		}
	}


	void game_of_life_cuda(unsigned char*& lifeData, const int worldWidth, const int worldHeight, unsigned char*& resultLifeData, int generations)
	{
		int blockSize = 256;
		int numBlocks = (blockSize + worldWidth * worldHeight - 1) / blockSize;
		for (int i = 0; i < generations; i++)
		{
			GameOfLife <<<numBlocks, blockSize>>> (lifeData, worldWidth, worldHeight, resultLifeData);
			hipDeviceSynchronize();
			std::swap(lifeData, resultLifeData);
		}
	}

	void init_world_cuda(unsigned char*& lifeData, int worldWidth, int worldHeight, unsigned char*& resultLifeData)
	{
		if (hipMalloc(&lifeData, (size_t)worldWidth * worldHeight * sizeof(unsigned char)) ||
			hipMalloc(&resultLifeData, (size_t)worldWidth * worldHeight * sizeof(unsigned char)))
		{
			printf("Allocation failed!");
		}

		int blockSize = 256;
		int numBlocks = (blockSize + worldWidth * worldHeight - 1) / blockSize;
		InitWorld <<<numBlocks, blockSize >>> (lifeData, worldWidth, worldHeight, resultLifeData);
		hipDeviceSynchronize();
	}

	void write_image_cuda(int imageWidth, int imageHeight, unsigned char* imageBuffer, int worldWidth, int worldHeight, unsigned char* lifeData)
	{
		int blockSize = 256;
		int numBlocks = (blockSize + imageWidth * imageHeight - 1) / blockSize;
		WriteImage <<<numBlocks, blockSize >>> (imageWidth, imageHeight, imageBuffer, worldWidth, worldHeight, lifeData);
		hipDeviceSynchronize();
	}

	void oscilator_detection(unsigned char*& lifeData, int worldWidth, int worldHeight, unsigned char*& resultLifeData, Pixel* image)
	{
		int blockSize = 256;
		int numBlocks = (blockSize + worldWidth * worldHeight - 1) / blockSize;
		GameOfLife <<<numBlocks, blockSize >>> (lifeData, worldWidth, worldHeight, resultLifeData);
		hipDeviceSynchronize();
		std::swap(lifeData, resultLifeData);
		WriteRGBImage <<<numBlocks, blockSize >>> (lifeData, worldWidth, worldHeight, image);
		hipDeviceSynchronize();
	}

	void init_world_with_image_cuda(int imageWidth, int imageHeight, unsigned char* imageBuffer, int worldWidth, int worldHeight,
		unsigned char* lifeData)
	{
		int blockSize = 256;
		int numBlocks = (blockSize + imageWidth * imageHeight - 1) / blockSize;
		InitWithImage <<<numBlocks, blockSize >>> (imageWidth, imageHeight, imageBuffer, worldWidth, worldHeight, lifeData);
		hipDeviceSynchronize();
	}
	
	__device__ bool isHorizontalBlinker(unsigned char* lifeData, int x, int yAbs, int xLeft, int xRight, int yAbsDown, int yAbsUp)
	{
		return lifeData[x + yAbs] == 1 &&
			lifeData[xLeft + yAbs] == 1 &&
			lifeData[xRight + yAbs] == 1 &&
			lifeData[xLeft + yAbsUp] == 0 &&
			lifeData[x + yAbsUp] == 0 &&
			lifeData[xRight + yAbsUp] == 0 &&
			lifeData[xLeft + yAbsDown] == 0 &&
			lifeData[x + yAbsDown] == 0 &&
			lifeData[xRight + yAbsDown] == 0;
	}

	__device__ bool isVerticalBlinker(unsigned char* lifeData, int x, int yAbs, int xLeft, int xRight, int yAbsDown, int yAbsUp)
	{
		return false;
	}
}
